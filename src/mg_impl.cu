#include "hip/hip_runtime.h"
#include "grid_size.hpp"

#include <iostream>

void checkForErrors(const hipError_t status, const char *label, const int line, const char *file)
{
    if (status != hipSuccess) {
        std::cerr << "CUDA ERROR (" << label << ") ";
        std::cerr << "at " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(status) << ". Exiting..." << std::endl;
        exit(1);
    }
}

//#define CUDA_CHECK_KERNEL_ERRORS

#define cuchck(func) checkForErrors(func, #func, __LINE__, __FILE__)
#ifdef CUDA_CHECK_KERNEL_ERRORS
    #warning !!! !!! !!! CUDA SYNC IS ACTIVE !!! !!! !!!
    #define cuchck_last() hipDeviceSynchronize(); cuchck(hipGetLastError())
#else
    #define cuchck_last()
#endif

float normoo_cu(const float* vec, unsigned int count) {
    // TODO
    return abs(1);
}

double normoo_cu(const double* vec, unsigned int count) {
    // TODO
    return abs(1);
}

float norm2_cu(const float* vec, unsigned int count) {
    return 1; // TODO
}

double norm2_cu(const double* vec, unsigned int count) {
    return 1; // TODO
}

template<typename T>
T norm_cu(const T* vec, unsigned int count) {
    return normoo_cu(vec, count);
}

namespace cuda {

    template<typename T>
    __global__
    void add(T* dst, const T* lhs, const T* rhs, const unsigned int count) {
        const int i (threadIdx.x + blockDim.x * blockIdx.x);
        if (i < count) {
            dst[i] = lhs[i] + rhs[i];
        }
    }

    template<typename T>
    __global__
    void sub(T* dst, const T* lhs, const T* rhs, const unsigned int count) {
        const int i (threadIdx.x + blockDim.x * blockIdx.x);
        if (i < count) {
            dst[i] = lhs[i] - rhs[i];
        }
    }
}

namespace cuda {
    template<typename T>
    __global__
    void zerof64(T* dst, const unsigned int count) {
        const int i (threadIdx.x + blockDim.x * blockIdx.x);
        if (i < count) {
            dst[i] = 0.0;
        }
    }

    template<typename T>
    __global__
    void addAssign(T* dst, const T* vec, const unsigned int count) {
        const int i (threadIdx.x + blockDim.x * blockIdx.x);
        if (i < count) {
            dst[i] += vec[i];
        }
    }

    //http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
    void reduce_sum() {
        std::cout << "ERR reduce sum" << std::endl;
    }
}


namespace cuda {
    /** x,y are block indices, i,j are thread indices in block x,y
     *  storage is row major: 00 01 02 03 ... 10 11 12 13 ... ...
     *
     *   o--> y,j
     *   |
     *   V x,i
     *
     *      +-------+-------+  \     ~.
     *      | 00 01 | 02 03 |   | Bx  |
     *      | 10 11 | 12 13 |   |     |
     *      +-------+-------+  /      | Gx
     *      | 20 21 | 22 23 |         |
     *      | 30 31 | 32 33 |         |
     *      +-------+-------+        ~'
     *
     *      \__By___/
     *
     *      ^*-----Gy------*^
     *
     *
     */
    template<typename T>
    __global__
    void residuum_neg_laplace(const T* u, const T* b, T* res, multi_index_t N, multi_real_t length) {
        const int i (threadIdx.x + blockIdx.x * blockDim.x);
        const int j (threadIdx.y + blockIdx.y * blockDim.y);
        const int ix (i + j * (N[0] + 2));

        if ((i <= N[0]+1 && j <= N[1]+1) && (i == 0 || j == 0 || i == N[0]+1 || j == N[1]+1)) {
            res[ix] = 0;
        } else if (i <= N[0] && j <= N[1]) {
            const int im1_j = ix - 1;
            const int ip1_j = ix + 1;
            const int i_jm1 = ix - (N[0] + 2);
            const int i_jp1 = ix + (N[0] + 2);
            res[ix] = b[ix] - (N[0]+1)*(N[1]+1)/length[0]/length[1]*(4*u[ix] - 1*u[im1_j] - 1*u[ip1_j] - 1*u[i_jm1] - 1*u[i_jp1]);
        }
    }

    template<typename T>
    __global__
    void jacobi_step_neg_laplace(double omega, const T* u0, T* u1, const T* b, multi_index_t N, multi_real_t length) {
        const int i (threadIdx.x + blockIdx.x * blockDim.x);
        const int j (threadIdx.y + blockIdx.y * blockDim.y);
        const int ix (i + j * (N[0] + 2));

        if ((i <= N[0]+1 && j <= N[1]+1) && (i == 0 || j == 0 || i == N[0]+1 || j == N[1]+1)) {
            // boundary is known
            u1[ix] = u0[ix];
        } else if (i <= N[0] && j <= N[1]) {
            const int im1_j = ix - 1;
            const int ip1_j = ix + 1;
            const int i_jm1 = ix - (N[0] + 2);
            const int i_jp1 = ix + (N[0] + 2);
            const float u_ = 0.25 * (b[ix] / ((N[0]+1)*(N[1]+1)) * length[0] * length[1] + u0[im1_j] + u0[ip1_j] + u0[i_jm1] + u0[i_jp1]);
            u1[ix] = (1-omega) * u0[ix] + omega * u_;
        }
    }

    // treads: N*N, one thread per per inner grid point
    template<typename T>
    __global__
    void jacobi_step_neg_laplace_inner(double omega, const T* u0, T* u1, const T* b, multi_index_t N, multi_real_t length) {
        const int i (1 + threadIdx.x + blockIdx.x * blockDim.x);
        const int j (1 + threadIdx.y + blockIdx.y * blockDim.y);
        const int ix (i + j * (N[0] + 2));

        if (i <= N[0] && j <= N[1]) {
            const int im1_j = ix - 1;
            const int ip1_j = ix + 1;
            const int i_jm1 = ix - (N[0] + 2);
            const int i_jp1 = ix + (N[0] + 2);
            const float u_ = 0.25 * (b[ix] / ((N[0]+1)*(N[1]+1)) * length[0] * length[1] + u0[im1_j] + u0[ip1_j] + u0[i_jm1] + u0[i_jp1]);
            u1[ix] = (1-omega) * u0[ix] + omega * u_;
        }
    }

    template<typename T>
    __global__
    void restrict_fw_2D(const T* v_N, T* v_n, multi_index_t N, multi_index_t n) {
        // big indices for 'N'-Matrices, small indices for 'n'-Matrices
        const int i (threadIdx.x + blockIdx.x * blockDim.x);
        const int j (threadIdx.y + blockIdx.y * blockDim.y);
        const int ix (i + j * (n[0] + 2));
        const int I (2*i);
        const int J (2*j);
        const int IX (I + J * (N[0] + 2));

        /*
            O . O . O
            . . . . .
            O . O . O
            . . . . .
            O . O . O
        */

        if ((i <= n[0]+1 && j <= n[1]+1) && (i == 0 || j == 0 || i == n[0]+1 || j == n[1]+1)) {
            // boundary is known
            v_n[ix] = v_N[IX];
        } else if (i <= n[0] && j <= n[1]) {
            const int dI = 1;
            const int dJ = N[0]+2;
            v_n[ix] = 1.0/16.0 * (
                1*v_N[IX-dI-dJ] + 2*v_N[IX-dJ] + 1*v_N[IX+dI-dJ] +
                2*v_N[IX-dI   ] + 4*v_N[IX   ] + 2*v_N[IX+dI   ] +
                1*v_N[IX-dI+dJ] + 2*v_N[IX+dJ] + 1*v_N[IX+dI+dJ]
            );
        }
    }

    template<typename T>
    __global__
    void interpolate_2D(const T* v_n, T* v_N, multi_index_t n, multi_index_t N) {
        // big indices for 'N'-Matrices, small indices for 'n'-Matrices
        const int i (threadIdx.x + blockIdx.x * blockDim.x);
        const int j (threadIdx.y + blockIdx.y * blockDim.y);
        const int ix (i + j * (N[0] + 2));
        const int I (2*i);
        const int J (2*j);
        const int IX (I + J * (N[0] + 2));

        /*
            O . O . O
            . . . . .
            O . O . O
            . . . . .
            O . O . O
        */

        if ((i <= n[0]+1 && j <= n[1]+1) && (i == 0 || j == 0)) {
            // boundary is known
            // no update assuming that the boundary of v_N is already set
        } else if (i <= n[0] && j <= n[1]) {
            /* working on patches like this when all '.' are inner points
                . .
                . O
            */
            const int di = 1;
            const int dj = n[0] + 2;
            const int dI = 1;
            const int dJ = N[0] + 2;
            v_N[IX] = v_n[ix];
            v_N[IX - dI] = 0.5 * (v_n[ix - di] + v_n[ix]);
            v_N[IX - dJ] = 0.5 * (v_n[ix - dj] + v_n[ix]);
            v_N[IX - dI - dJ] = 0.25 * (v_n[ix - di] + v_n[ix - dj] + v_n[ix - di - dj] + v_n[ix]);
        } else if (i <= n[0]+1 && j <= n[1]+1) {
            /* working on patches like this when some '.' are bondary points

                . .
                . O
                ----

                . . |
                . O |

                . . |
                . O |
                ----+
            */
            const int di = 1;
            const int dj = n[0] + 2;
            const int dI = 1;
            const int dJ = N[0] + 2;
            // don't do v_N[IX] = v_n[ix]; as the point is on the border for sure -> no update
            if (J != N[1]+1) v_N[IX - dI] = 0.5 * (v_n[ix - di] + v_n[ix]);
            if (I != N[0]+1) v_N[IX - dJ] = 0.5 * (v_n[ix - dj] + v_n[ix]);
            v_N[IX - dI - dJ] = 0.25 * (v_n[ix - di] + v_n[ix - dj] + v_n[ix - di - dj] + v_n[ix]);
        }


    }

    // treads N*N: one thrad per inner grid point
    template<typename T>
    __global__
    void addAssign2Dinner(T* dst, const T* vec, multi_index_t N) {
        // index 0 is on the border
        const int i (1 + threadIdx.x + blockIdx.x * blockDim.x);
        const int j (1 + threadIdx.y + blockIdx.y * blockDim.y);
        const int ix (i + j * (N[0] + 2));
        if (i < N[0]+1 && j < N[1]+1) {
            dst[ix] += vec[ix];
        }
    }

}

template<typename T>
struct Fn_CUDA_mem {
    static bool mem_device_host_equal() {
        return false;
    }

    static T* malloc_typed(unsigned int count) {
        T* ptr = 0;
        cuchck(hipMalloc(&ptr, sizeof(T) * count));
        return ptr;
    }

    static void free_typed(T* ptr) {
        cuchck(hipFree(ptr));
    }

    static void memcpy_typed_HostToDevice(T* dst, const T* src, unsigned int count) {
        cuchck(hipMemcpy(dst, src, sizeof(T) * count, hipMemcpyHostToDevice));
    }

    static void memcpy_typed_DeviceToHost(T* dst, const T* src, unsigned int count) {
        cuchck(hipMemcpy(dst, src, sizeof(T) * count, hipMemcpyDeviceToHost));
    }

    static void memcpy_typed_DeviceToDevice(T* dst, const T* src, unsigned int count) {
        cuchck(hipMemcpy(dst, src, sizeof(T) * count, hipMemcpyDeviceToDevice));
    }

    static void zero(T* dst, const unsigned int count) {
        dim3 block(1024, 1, 1);
        dim3 grid((unsigned)ceil(count/(double)block.x), 1, 1);
        cuda::zerof64<<<grid, block>>>(dst, count);
        cuchck_last();
    }
};

template<typename T>
struct Fn_laplace_cuda : Fn_CUDA_mem<T>, Grid2D {
    static void restrict(multi_index_t N, const T* v_N, T* v_n) {
        const multi_index_t n = coarsen(N);
        dim3 block(1, 512, 1);
        dim3 grid((unsigned)ceil((N[0]+2)/(double)block.x), (unsigned)ceil((N[1]+2)/(double)block.y), 1);
        cuda::restrict_fw_2D<<<grid, block>>>(v_N, v_n, N, n);
        cuchck_last();
    }

    static void interpolate(multi_index_t n, const T* v_n, T* v_N, const char* mask) {
        const multi_index_t N (n[0]*2 + 1, n[1]*2 + 1);
        dim3 block(1, 512, 1);
        dim3 grid((unsigned)ceil((N[0]+2)/(double)block.x), (unsigned)ceil((N[1]+2)/(double)block.y), 1);
        cuda::interpolate_2D<<<grid, block>>>(v_n, v_N, n, N);
        cuchck_last();
    }

    static void residuum(multi_index_t N, multi_real_t length, const T* u, const T* b, const char* mask, T* res) {
        dim3 block(2, 64, 1);
        dim3 grid((unsigned)ceil((N[0]+2)/(double)block.x), (unsigned)ceil((N[1]+2)/(double)block.y), 1);
        cuda::residuum_neg_laplace<<<grid, block>>>(u, b, res, N, length);
        cuchck_last();
    }

    static T norm(multi_index_t N, const T* vec0) {
        return ::norm_cu(vec0, size_N(N));
    }

    static T norm_residuum(multi_index_t N, multi_real_t length, const T* u, const T* b, const char* mask, T* scratch) {
      residuum(N, length, u, b, mask, scratch);
      return norm_cu(scratch, size_N(N));
    }

    // TODO scratch buffer might be obsolate
    static unsigned int _jacobi(unsigned int max_iters, double max_r, double omega, T* u0, T* u1, const T* b, multi_index_t N, multi_real_t length, const char* mask, T* scratch) {
        // as we copy (only if we have to) the result back to u0, which has its
        // initialized, we can also only interate on the inner points, if
        // we initialize the u1 borders first

        dim3 block(1, 512, 1);
        dim3 grid_all  ((unsigned)ceil((N[0]+2)/(double)block.x), (unsigned)ceil((N[1]+2)/(double)block.y), 1);
        dim3 grid_inner((unsigned)ceil( N[0]   /(double)block.x), (unsigned)ceil( N[1]   /(double)block.y), 1);

        T * const dst = u0;
        unsigned int iters = 1;
        T r = norm_residuum(N, length, u0, b, mask, u1);
        while (iters <= max_iters && r >= max_r) {
            T r_old = r;

            // as mcpy will cause a sync, we spend one more block in the first iteration
            // to initialize the borders in u1
            if (iters == 1) {
                cuda::jacobi_step_neg_laplace<<<grid_all, block>>>(omega, u0, u1, b, N, length);
            } else {
                cuda::jacobi_step_neg_laplace_inner<<<grid_inner, block>>>(omega, u0, u1, b, N, length);
            }
            cuchck_last();


            // swap input and output
            std::swap(u0, u1);

            r = norm_residuum(N, length, u0, b, mask, scratch);
            ++iters;

            if (r == r_old) {
                if (max_r != 0) break; // max_res == 0 means no residuum checks
            }
        }
        if (dst != u0) {
            hipMemcpy(dst, u0, size_N(N) * sizeof(T), hipMemcpyDeviceToDevice);
        }
        return iters - 1;
    }

    static unsigned int solve(unsigned int max_iters, double max_r, multi_index_t N, multi_real_t length, T* u0, T* u1, const T* b, const char* mask, T* scratch) {
        return _jacobi(max_iters, max_r, 1.0, u0, u1, b, N, length, mask, scratch);
    }

    static unsigned int smooth(unsigned int max_iters, double max_r, multi_index_t N, multi_real_t length, T* u0, T* u1, const T* b, const char* mask, T* scratch) {
        return _jacobi(max_iters, max_r, 4.0/5.0, u0, u1, b, N, length, mask, scratch);
    }

    static double norm_sub(multi_index_t N, const T* vec0, const T* vec1, T* scratch) {
        dim3 block(1, 512, 1);
        dim3 grid((unsigned)ceil((N[0]+2)/(double)block.x), (unsigned)ceil((N[1]+2)/(double)block.y), 1);
        cuda::sub<<<grid, block>>>(scratch, vec0, vec1, size_N(N));
        cuchck_last();
        return norm_cu(scratch, size_N(N));
    }

    static void add_correction(multi_index_t N, T* u, const T* e, const char* mask) {
        dim3 block(1, 512, 1);
        dim3 grid((unsigned)ceil(N[0]/(double)block.x), (unsigned)ceil(N[1]/(double)block.y), 1);
        cuda::addAssign2Dinner<<<grid, block>>>(u, e, N); //update only inner points
        cuchck_last();
    }
};

// explicit instantiations
template struct Fn_CUDA_mem<float>;
template struct Fn_CUDA_mem<double>;
template struct Fn_laplace_cuda<float>;
template struct Fn_laplace_cuda<double>;
